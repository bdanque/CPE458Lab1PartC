#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	 float result = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
	 int i = 0; int k =0;
	
    //get memory needed for this block
    __shared__ float N_shared[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float M_shared[BLOCK_SIZE][BLOCK_SIZE];

	 //increment THIS tile from edge to edge of M...
    int stop = M.width % BLOCK_SIZE ? (M.width/BLOCK_SIZE) + 1 : M.width/BLOCK_SIZE;
    for (i = 0; i <  stop ; i++)
    {
        //transfer to shared mem        
        M_shared[threadIdx.y][threadIdx.x] = M.elements[row * M.width +
            (i * BLOCK_SIZE + threadIdx.x)];
        N_shared[threadIdx.y][threadIdx.x] = N.elements[(i * BLOCK_SIZE + threadIdx.y)
            * N.width + col];
		  
		  //regroup after caching
        __syncthreads();  

        for(k = 0; (k < BLOCK_SIZE) && ((k + i*BLOCK_SIZE) < (M.width)); k++)
        {
            //rolling sum
            result += M_shared[threadIdx.y][k] * N_shared[k][threadIdx.x];
        }

        //regroup
        __syncthreads();
    }
    P.elements[row * P.width + col] = (float) result;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
